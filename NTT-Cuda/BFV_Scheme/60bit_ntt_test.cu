#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"
#include "ntt_60bit.cuh"
#include "poly_arithmetic.cuh"

#include <time.h>
#include <sys/time.h> 

#define check 0

int main()
{
    unsigned N = 4096 << 1;
    // unsigned N = 32768;

    int size_array = sizeof(unsigned long long) * N;
    int size = sizeof(unsigned long long);

    unsigned long long q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams(q, psi, psiinv, ninv, q_bit, N);

    unsigned long long* psiTable = (unsigned long long*)malloc(size_array);
    unsigned long long* psiinvTable = (unsigned long long*)malloc(size_array);
    fillTablePsi128(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi
    // exit(0);

    //copy powers of psi and psi inverse tables to device
    unsigned long long* psi_powers, * psiinv_powers;

    hipMalloc(&psi_powers, size_array);
    hipMalloc(&psiinv_powers, size_array);

    hipMemcpy(psi_powers, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(psiinv_powers, psiinvTable, size_array, hipMemcpyHostToDevice);

    // cout << "n = " << N << endl;
    // cout << "q = " << q << endl;
    // cout << "Psi = " << psi << endl;
    // cout << "Psi Inverse = " << psiinv << endl;

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    uint128_t mu1 = uint128_t::exp2(bit_length * 2);
    unsigned long long mu = (mu1 / q).low;
    
    clock_t start,stop;
  	start = clock();
    // MyTimer timer;
    // MyTimer timer2;
    // long ntt_time = 0,intt_time = 0, trans_in_time = 0, trans_out_time = 0;
    // timer2.Start();
    unsigned long long* a;
    hipHostMalloc(&a, sizeof(unsigned long long) * N);
    // randomArray128(a, N, q);
    unsigned long long* b;  
    hipHostMalloc(&b, sizeof(unsigned long long) * N);
    for(int j = 0; j < 100000; ++j)
    {
        // timer.Start();
        
        // randomArray128(b, N, q);
        for (size_t i = 0; i < N; i++)
        {
            a[i] = i;
            b[i] = 0;
        }
        b[0] = 1;
        unsigned long long* d_a;
        hipMalloc(&d_a, size_array);
        unsigned long long* d_b;
        hipMalloc(&d_b, size_array);
        hipStream_t ntt1, ntt2;
        // hipStream_t ntt1;
        hipStreamCreate(&ntt1);
        hipStreamCreate(&ntt2);
        hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
        hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);
        hipDeviceSynchronize();
        // timer.End();
        // trans_in_time += timer.costTime;
        // timer.Reset();

        // timer.Start();
        // forwardNTTdouble(d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers);
        forwardNTT(d_a, N, ntt1, q, mu, bit_length, psi_powers);
        hipDeviceSynchronize();
        // timer.End();
        // std::cout << "NTT-time:" << timer.costTime << "us" << std::endl;
        // ntt_time += timer.costTime;
        // timer.Reset();
        forwardNTT(d_b, N, ntt1, q, mu, bit_length, psi_powers);
        
        barrett << <N / 256, 256 >> > (d_a, d_b, q, mu, bit_length);
        // timer.Start();
        inverseNTT(d_a, N, ntt1, q, mu, bit_length, psiinv_powers);
        // hipDeviceSynchronize();
        // timer.End();
        // std::cout << "INTT-time:" << timer.costTime << "us" << std::endl;
        // intt_time += timer.costTime;
        // timer.Reset();

        // timer.Start();
        hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, 0);
        hipDeviceSynchronize();
        // timer.End();
        // std::cout << "INTT-time:" << timer.costTime << "us" << std::endl;
        // trans_out_time += timer.costTime;
        // timer.Reset();

        hipStreamDestroy(ntt1); 
        // hipStreamDestroy(ntt2);
        // timer2.End();
        // for (size_t i = 0; i < N; i++)
        // {
        //     cout << a[i] << " " ;
        // }
        // cout << endl;
        // hipHostFree(a); 
        // hipHostFree(b);
    }
    // timer2.End();
    // std::cout << "trans_in-time:" <<trans_in_time/10000 << "us" << std::endl;
    // std::cout << "trans_out-time:" << trans_out_time/10000 << "us" << std::endl;
    // std::cout << "NTT-time:" << ntt_time/1 << "us" << std::endl;
    // std::cout << "INTT-time:" << intt_time/1 << "us" << std::endl;
    // std::cout << "all-time:" << timer2.costTime/1 << "us" << std::endl;
    
	stop = clock();
  	double endtime=(double)(stop-start)/CLOCKS_PER_SEC; 
 	std::cout << "time: "<< endtime << "s" <<std::endl;

    // unsigned long long* a;
    // hipHostMalloc(&a, sizeof(unsigned long long) * N);
    // // randomArray128(a, N, q); //fill array with random numbers between 0 and q - 1
    

    // unsigned long long* b;  
    // hipHostMalloc(&b, sizeof(unsigned long long) * N);
    // // randomArray128(b, N, q); //fill array with random numbers between 0 and q - 1
    // for (size_t i = 0; i < N; i++)
    // {
    //     a[i] = i;
    //     b[i] = 0;
    // }
    // // for (size_t i = 0; i < 4; i++)
    // // {
    // //     a[i] = 1;
    // // }
    
    // b[0] = 1;
    

    // unsigned long long* d_a;
    // hipMalloc(&d_a, size_array);
    // unsigned long long* d_b;
    // hipMalloc(&d_b, size_array);

    // unsigned long long* refc;
    // if (check)
    //     refc = refPolyMul128(a, b, q, N);

    // hipStream_t ntt1, ntt2;
    // hipStreamCreate(&ntt1);
    // hipStreamCreate(&ntt2);

    // hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
    // hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);

    // // forwardNTT(d_a, N, ntt1, q, mu, bit_length, psi_powers);
    // // cout << "mu1.low = " << mu1.low << "mu1.high = " << mu1.high << "; mu = " << mu << endl;
    // forwardNTTdouble(d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers);
    // // forwardNTT(d_b, N, ntt1, q, mu, bit_length, psi_powers);
    
    // barrett << <N / 256, 256 >> > (d_a, d_b, q, mu, bit_length);
    // inverseNTT(d_a, N, ntt1, q, mu, bit_length, psiinv_powers);

    // hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, 0);

    // hipDeviceSynchronize();

    // hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);

    // // for (size_t i = 0; i < N; i++)
    // // {
    // //     cout << a[i] << " " ;
    // // }
    

    // // if (check) //check the correctness of results
    // // {
    // //     for (int i = 0; i < N; i++)
    // //     {
    // //         if (a[i] != refc[i])
    // //         {
    // //             cout << "error" << endl;
    // //             cout << i << "   " << a[i] << "   " << refc[i] << endl;
    // //         }

    // //     }

    // //     free(refc);
    // // }

    // hipHostFree(a); hipHostFree(b);

    return 0;
}


